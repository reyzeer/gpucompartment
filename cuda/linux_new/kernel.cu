#include "hip/hip_runtime.h"
//CUDA
#include "hip/hip_runtime.h"
#include ""
#include <thrust/iterator/counting_iterator.h>

//Others
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <ctime>

using namespace std;

typedef unsigned long long int _uint64;

#define THREADS 512

/* ------------------------------------------------------------
 	 	 	 CUDA func
 ------------------------------------------------------------ */

__device__ bool isPrime;

__global__ void primeNumberTesting(_uint64 number, _uint64 maxTestNumber) {

//__host__ __device__ void primeNumberTesting(_uint64 number, _uint64 maxTestNumber) {

	_uint64 threads	= blockDim.x;	//liczba watkow
	_uint64 thread	= threadIdx.x;	//numer aktualnego watku
	//_uint64 round	= blockIdx.x;	//numer przebiegu petli

	int inc = threads * 2;
	_uint64 n = (thread + 1) * 2 + 1;
	n = (n == 1) ? 2 : n;
	for ( ; n < maxTestNumber; n += inc) {
		if (!isPrime) {
			return;
		}
		if (number % n == 0) {
			isPrime = false;
			return;
		}
	
	}

}

/*__global__ void lackOfdivisorsInRange(_uint64 iNumber, _uint64 iStartRange, _uint64 iEndRange) {

    _uint64 threads	= blockDim.x;    //liczba watkow
    _uint64 thread	= threadIdx.x;   //numer aktualnego watku
    _uint64 round	= blockIdx.x;    //numer przebiegu petli

    //sprawdzany dzielnik
    _uint64 n = ((thread * 2 + 1) + 2 * threads * round) + iStartRange - ((iStartRange%2==1) ? 1 : 0);

    if (n <= iEndRange) {
        if (iNumber % n == 0) {
            isPrime = false;
        }
    }

    __syncthreads();

}*/



/* ------------------------------------------------------------
 	 	 	 	 	 	 CPU func
 ------------------------------------------------------------ */

bool primeNumberTestingStart(_uint64 number) {

	_uint64 iMaxTestNumber = sqrt((double) number); //maksymalna sprawdza wartosc

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	bool isPrime_Host = true;

	hipMemcpyToSymbol(HIP_SYMBOL(isPrime),&isPrime_Host,sizeof(bool),0,hipMemcpyHostToDevice);

	primeNumberTesting<<<1, THREADS>>>(number, iMaxTestNumber);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to launch primeNumberTesting kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpyFromSymbol(&isPrime_Host,HIP_SYMBOL(isPrime),sizeof(bool),0,hipMemcpyDeviceToHost);

	return isPrime_Host;

}

/*bool lackOfdivisorsInRange(_uint64 iNumber, _uint64 iStartRange, _uint64 iEndRange) {

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	bool isPrime_Host = true;

	hipMemcpyToSymbol(HIP_SYMBOL(isPrime),&isPrime_Host,sizeof(bool),0,hipMemcpyHostToDevice);

	_uint64 blocksPerGrid = (iEndRange - iStartRange + 1)/THREADS/2+1;
	primeNumberTesting<<<blocksPerGrid, THREADS>>>(_uint64 iNumber, _uint64 iStartRange, _uint64 iEndRange);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to launch primeNumberTesting kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpyFromSymbol(&isPrime_Host,HIP_SYMBOL(isPrime),sizeof(bool),0,hipMemcpyDeviceToHost);

	return isPrime_Host;

}*/

/* ------------------------------------------------------------
 main
 ------------------------------------------------------------ */

void mersensNumberTest()
{

	//Tablica liczb Mersena
	_uint64 mersensNumber[32];
	for (int j = 0; j < 32; j++) {
		_uint64 number = 1;
		for (_uint64 i = 0; i < j; number *= 2, i++);
		number--;
		mersensNumber[j] = number;
	}

	cout << "Start.\n";
	clock_t start = clock();
	for (int j = 0; j < 1000; j++) {
		for (int i = 0; i < 32; i++) {
			cout << mersensNumber[i] << " ";
			if (primeNumberTestingStart(mersensNumber[i])) {
				cout << "jest pierwsza.\n";
			}
			else {
				cout << "nie jest pierwsza.\n";
			}
		}
	}
	printf( "Czas wykonywania: %f ms\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void all32bitsNumber()
{
	
	cout << "Start.\n";
	clock_t start = clock();
	for (int i = 2; i < 4294967296; i++) {
		
		primeNumberTestingStart(i);

		/*cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}*/

	}
	printf( "Czas wykonywania: %f ms\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void all24bitsNumber()
{
	
	cout << "Start.\n";
	clock_t start = clock();
	for (int i = 2; i < 16777216; i++) {
		
		primeNumberTestingStart(i);

		/*cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}*/

	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void all16bitsNumber()
{
	
	cout << "Start.\n";
	clock_t start = clock();
	for (int i = 2; i < 65536; i++) {
		
		primeNumberTestingStart(i);

		/*cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}*/

	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void rangeTest(_uint64 startRange, _uint64 endRange)
{

	if (endRange >= 18446744073709551615) {
		endRange = 18446744073709551614;
	}

	cout << "Start.\n";
	clock_t start = clock();
	for (_uint64 i = startRange; i <= endRange; i++) {
		
		/*if (primeNumberTestingStart(i)) {
			cout << i << " jest pierwsza.\n";
		}*/

		clock_t start2 = clock();
		cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}
		printf( "Czas wykonywania: %f s\n", ((double) clock() - start2)/CLOCKS_PER_SEC );

	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

int main()
{

	/*clock_t start = clock();
	if (primeNumberTestingStart(821)) {
		cout << "jest pierwsza.\n";
	}
	else {
		cout << "nie jest pierwsza.\n";
	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

	start = clock();
	if (primeNumberTestingStart(820)) {
		cout << "jest pierwsza.\n";
	}
	else {
		cout << "nie jest pierwsza.\n";
	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

	start = clock();
	if (primeNumberTestingStart(18446744073709551614-3)) {
		cout << "jest pierwsza.\n";
	}
	else {
		cout << "nie jest pierwsza.\n";
	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );*/

	//mersensNumberTest()
	//all16bitsNumber();
	//all24bitsNumber();
	//all32bitsNumber();

	_uint64 endRange = 18446744073709551614;
	_uint64 startRange = endRange - 256;
	//for (int i = 0; i < 10; i++) {
		rangeTest(startRange, endRange);
	//}

	return 0;

}
