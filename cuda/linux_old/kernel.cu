#include "hip/hip_runtime.h"
//CUDA
#include "hip/hip_runtime.h"
#include ""
#include <thrust/iterator/counting_iterator.h>

//Others
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <ctime>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

typedef unsigned long long int _uint64;

#define THREADS 1024

/* ------------------------------------------------------------
 	 	 	 CUDA func
 ------------------------------------------------------------ */

__device__ bool isPrime;

__global__ void primeNumberTesting(_uint64 iNumber, _uint64 iMaxTestNumber) {

	if (!isPrime) {
		printf("trap\n");
		__threadfence();
		//return;		
		asm("trap;");
	}

	_uint64 threads	= blockDim.x;	//liczba watkow
	_uint64 thread	= threadIdx.x;	//numer aktualnego watku
	_uint64 round	= blockIdx.x;	//numer przebiegu petli

	_uint64 n = (((thread * 2 + 1) + 2 * threads * round) == 1)		//sprawdzany dzielnik
			? 2 : ((thread * 2 + 1) + 2 * threads * round);
	
	if (n <= iMaxTestNumber) {
		if (iNumber % n == 0) {
			isPrime = false;
			
			//*status = 0;
			//__threadfence_system();
			//asm("trap;"); //anuluje wszystkie watki	
			
			return;

			//przerywania dzialanie
			//__threadfence();
			//asm("trap;"); //anuluje wszystkie watki
			//asm("exit;"); //anuluje ten watek
		}
	}

}

__global__ void fermatPrimeNumberTest(_uint64 number, _uint64 k) {

	if (!isPrime) {
		return;		
	}

	//_uint64 threads	= blockDim.x;	//liczba watkow
	_uint64 thread	= threadIdx.x;	//numer aktualnego watku
	_uint64 round	= blockIdx.x;	//numer przebiegu petli

	if (thread * round < k) {

		  hiprandState_t state;

		  /* we have to initialize the state */
		  hiprand_init(0, /* the seed controls the sequence of random values that are produced */
			      0, /* the sequence number is only important with multiple cores */
			      0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
			      &state);

		  /* hiprand works like rand - except that it takes a state as a parameter */

		_uint64  i, random, x;
		random = ( hiprand(&state) % (number-1) ) + 1;
		x = ((_uint64) powf( (float) random, number - 1));
		if (x % random != 1) {
			isPrime = false;
			return;
		}
	}

}

/*__global__ void lackOfdivisorsInRange(_uint64 iNumber, _uint64 iStartRange, _uint64 iEndRange) {

    _uint64 threads	= blockDim.x;    //liczba watkow
    _uint64 thread	= threadIdx.x;   //numer aktualnego watku
    _uint64 round	= blockIdx.x;    //numer przebiegu petli

    //sprawdzany dzielnik
    _uint64 n = ((thread * 2 + 1) + 2 * threads * round) + iStartRange - ((iStartRange%2==1) ? 1 : 0);

    if (n <= iEndRange) {
        if (iNumber % n == 0) {
            isPrime = false;
        }
    }

    __syncthreads();

}*/



/* ------------------------------------------------------------
 	 	 	 	 	 	 CPU func
 ------------------------------------------------------------ */

bool primeNumberTestingStart(_uint64 number) {

	_uint64 iMaxTestNumber = sqrt((double) number); //maksymalna sprawdza wartosc

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	bool isPrime_Host = true;

	hipMemcpyToSymbol(HIP_SYMBOL(isPrime),&isPrime_Host,sizeof(bool),0,hipMemcpyHostToDevice);

	_uint64 blocksPerGrid = iMaxTestNumber/THREADS/2+1;
	
	cout << number << endl;
	primeNumberTesting<<<blocksPerGrid, THREADS>>>(number, iMaxTestNumber);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch primeNumberTesting kernel (error code %s)!\n",
				hipGetErrorString(err));
		//exit(EXIT_FAILURE);
	}

	isPrime_Host = false;

	//cout << "?: " << isPrime_Host << endl;
	//cout << "??: " << isPrime << endl;

	hipMemcpyFromSymbol(&isPrime_Host,HIP_SYMBOL(isPrime),sizeof(bool),0,hipMemcpyDeviceToHost);

	return isPrime_Host;

}

_uint64 fermatPrimeNumberTestStart(_uint64 number, _uint64 k) {

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	bool isPrime_Host = true;

	hipMemcpyToSymbol(HIP_SYMBOL(isPrime),&isPrime_Host,sizeof(bool),0,hipMemcpyHostToDevice);

	_uint64 blocksPerGrid = k/THREADS+1;

	primeNumberTesting<<<blocksPerGrid, THREADS>>>(number, k);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch primeNumberTesting kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpyFromSymbol(&isPrime_Host,HIP_SYMBOL(isPrime),sizeof(bool),0,hipMemcpyDeviceToHost);

	if (isPrime_Host) {
		return k;
	}
	else {
		return 0;
	}

}

/*bool lackOfdivisorsInRange(_uint64 iNumber, _uint64 iStartRange, _uint64 iEndRange) {

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	bool isPrime_Host = true;

	hipMemcpyToSymbol(HIP_SYMBOL(isPrime),&isPrime_Host,sizeof(bool),0,hipMemcpyHostToDevice);

	_uint64 blocksPerGrid = (iEndRange - iStartRange + 1)/THREADS/2+1;
	primeNumberTesting<<<blocksPerGrid, THREADS>>>(_uint64 iNumber, _uint64 iStartRange, _uint64 iEndRange);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to launch primeNumberTesting kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipMemcpyFromSymbol(&isPrime_Host,HIP_SYMBOL(isPrime),sizeof(bool),0,hipMemcpyDeviceToHost);

	return isPrime_Host;

}*/

/* ------------------------------------------------------------
 main
 ------------------------------------------------------------ */

void mersensNumberTest()
{

	//Tablica liczb Mersena
	_uint64 mersensNumber[32];
	for (int j = 0; j < 32; j++) {
		_uint64 number = 1;
		for (_uint64 i = 0; i < j; number *= 2, i++);
		number--;
		mersensNumber[j] = number;
	}

	cout << "Start.\n";
	clock_t start = clock();
	for (int j = 0; j < 1000; j++) {
		for (int i = 0; i < 32; i++) {
			cout << mersensNumber[i] << " ";
			if (primeNumberTestingStart(mersensNumber[i])) {
				cout << "jest pierwsza.\n";
			}
			else {
				cout << "nie jest pierwsza.\n";
			}
		}
	}
	printf( "Czas wykonywania: %f ms\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void all32bitsNumber()
{
	
	cout << "Start.\n";
	clock_t start = clock();
	for (int i = 2; i < 4294967296; i++) {
		
		primeNumberTestingStart(i);

		/*cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}*/

	}
	printf( "Czas wykonywania: %f ms\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void all24bitsNumber()
{
	
	cout << "Start.\n";
	clock_t start = clock();
	for (int i = 2; i < 16777216; i++) {
		
		primeNumberTestingStart(i);

		/*cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}*/

	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void all16bitsNumber()
{
	
	cout << "Start.\n";
	clock_t start = clock();
	for (int i = 2; i < 65536; i++) {
		
		primeNumberTestingStart(i);

		/*cout << i << " ";
		if (primeNumberTestingStart(i)) {
			cout << "jest pierwsza.\n";
		}
		else {
			cout << "nie jest pierwsza.\n";
		}*/

	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

void rangeTest(_uint64 startRange, _uint64 endRange)
{

	if (endRange >= 18446744073709551615) {
		endRange = 18446744073709551614;
	}

	cout << "Start.\n";
	clock_t start = clock();
	for (_uint64 i = startRange; i <= endRange; i++) {
		if (primeNumberTestingStart(i)) {
			cout << i << " jest pierwsza.\n";
		}
	}
	printf( "Czas wykonywania: %f s\n", ((double) clock() - start)/CLOCKS_PER_SEC );

}

int main()
{

	srand(time(NULL));

	cout << fermatPrimeNumberTestStart(47, 10) << endl;
	cout << fermatPrimeNumberTestStart(821, 10) << endl;

	return 0;

	//mersensNumberTest()
	//all16bitsNumber();
	//all24bitsNumber();
	//all32bitsNumber();

	_uint64 endRange = 18446744073709551614;
	_uint64 startRange = endRange - 256;
	//for (int i = 0; i < 10; i++) {
		rangeTest(startRange, endRange);
	//}

	return 0;

}
